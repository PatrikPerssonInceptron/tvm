#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

namespace {

int64_t SecondsToCycles(const double seconds) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  return int64_t(seconds * double(prop.clockRate) * 1000.0);
}

__global__ void BusyWaitKernel(const int64_t num_cycles) {
  // Based on pytorch/aten/src/ATen/cuda/Sleep.cu::spin_kernel
  const auto start = clock64();
  for (int64_t clock_delta = 0; clock_delta < num_cycles; clock_delta = clock64() - start);
}

}  // namespace

namespace tvm {
namespace runtime {
namespace cuda {

void BusyWait(const double duration, ihipStream_t* stream) {
  const auto num_cycles = SecondsToCycles(duration);
  BusyWaitKernel<<<1, 1, 0, stream>>>(num_cycles);
}

}  // namespace cuda
}  // namespace runtime
}  // namespace tvm
